
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS_NUM 1
#define THREADS_PER_BLOCK 4

__global__ void hello_from_gpu() {
    int thread_id = threadIdx.x;
    printf("Thread %d!\n", thread_id);
}

int main() {
    hello_from_gpu<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();
    return 0;
}
